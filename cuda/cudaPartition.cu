/*
 * cudaPartition.cu
 * Solve the Partition problem using CUDA.
 * https://en.wikipedia.org/wiki/Partition_problem
 * This code works for multisets of up to 32 elements
 * The input is expected to be as follows:
 * The first line will contain n, the number of elements in the multiset
 * The remaining n lines will contain the n values, one per line
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// The kernel
// This function is executed, in parallel, on the processors on the GPU card
//
__global__
void evaluatePartition(  int n, int *array,int *result) {
  unsigned int value = blockIdx.x*blockDim.x + threadIdx.x;
  int sum0s = 0;
  int sum1s = 0;
  unsigned int mask = 1;
  for(int i = 0;i < n;i++) {
    if ((mask & value) != 0) {
      sum1s = sum1s + array[i];
    }
    else {
      sum0s = sum0s + array[i];
    }
    mask = mask * 2;
  }
  if (sum0s == sum1s)
     result[value] = 1;
  else
     result[value] = 0;
  // printf("%d %d \n",value,result[value]);
}

void printResults(unsigned int value,int n,int *array) 
{
  printf("Solution:\n");
  printf("First partition: ") ;
  unsigned int mask = 1;
  int sum = 0;
  for(int i = 0;i < n;i++) {
    if ((mask & value) != 0) {
      printf("%d ",array[i]);
      sum = sum + array[i];
    }
    mask = mask * 2;
  }
  printf(" sum: %d \n",sum);
  printf("Second partition: ") ;
  mask = 1;
  sum = 0;
  for(int i = 0;i < n;i++) {
    if ((mask & value) == 0) {
      printf("%d ",array[i]);
      sum = sum + array[i];
    }
    mask = mask * 2;
  }
  printf(" sum: %d \n",sum);
}


int main() {

  int n;
  int *array;

  scanf("%d",&n);

  printf("The value of n is %d\n",n);
  array = (int *) malloc (n * sizeof(int));
  for(int i = 0;i < n;i++) {
    scanf("%d",&array[i]);
  }
  printf("The read values are: \n");
  for(int i = 0;i < n;i++) {
    printf("%d ",array[i]);
  }
  printf("\n");

  unsigned int nPartitions = 1;
  for(int i = 0;i < n;i++) {
    nPartitions = nPartitions * 2;
  }
  // printf("The number of possible partitions is: %d\n",nPartitions);
  // Only half of all possible partitions need be examined
  // The second half is symmetrical to the first half
  nPartitions = nPartitions / 2;

  int solutionFound = 0;
  int solution = -1;
  // Allocate the variables in the device:
  // The array with the integer values in the device is called d_array
  int *d_array;
  hipMalloc(&d_array, n*sizeof(int));

  // Copy the variables from the host to the device
  hipMemcpy(d_array,array,n*sizeof(int), hipMemcpyHostToDevice);
 
  // Allocate on the device an array to keep all the results
  int *d_results;
  hipMalloc(&d_results,nPartitions*sizeof(int));
// Now invoke the kernel
  evaluatePartition<<<(nPartitions+31)/32,32>>>(  n, d_array,d_results) ;
  // The array on the host that will contain the results is called results
  int *results;
  results = (int *) calloc (nPartitions , sizeof(int));
// Copy the results from the GPU card to main memory on the host
  hipMemcpy(results,d_results,nPartitions*sizeof(int),hipMemcpyDeviceToHost);
  /*
  for(int i = 0;i < nPartitions;i++) {
	 printf("%d ",results[i]);
  }
  printf("\n");
 */ 
  for(int i = 0;i < nPartitions;i++) {
	  if (results[i] != 0) {
		  solutionFound = 1;
		  solution = i;
		  break;
	  }
  }
   
  if (solutionFound == 1) {
    printResults(solution, n, array);
  }
  else {
    printf("No solution was found.");
  }
  return 0;
}
 
